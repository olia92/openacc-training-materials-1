/*
 * Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

extern "C" void dot_acc(int*, int*, int*, int, int);
extern "C" void dot(int*, int*, int*, int, int);
extern "C" void map(int*, int*, int);
extern "C" void unmap(int*);

int main()
{

	int i, j, m, n;
	int *A, *B, *C, *D;
	int *A_d, *B_d, *C_d;

	srand(0);

	m = 4098;
	n = 4098;

	A = (int*) malloc( m*n * sizeof(int));
	B = (int*) malloc( m*n * sizeof(int));
	C = (int*) malloc(  m  * sizeof(int));
	D = (int*) malloc(  m  * sizeof(int));

	for( i = 0; i < m; i++ ) {
		for( j = 0; j < n; j++ ) {
			A[i*n+j] = rand() % 100 + 1;
			B[i*n+j] = rand() % 100 + 1;
		}
	}

	hipMalloc((void **)&A_d, m*n*sizeof(int));
	hipMalloc((void **)&B_d, m*n*sizeof(int));
	hipMalloc((void **)&C_d, m*  sizeof(int));
	
	hipMemcpy(A_d, A, m*n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(B_d, B, m*n*sizeof(int), hipMemcpyHostToDevice);

	map(A, A_d, m*n*sizeof(int));
	map(B, B_d, m*n*sizeof(int));
	map(C, C_d, m*sizeof(int));

	dot_acc(A,B,C,m,n);
	
	hipMemcpy(C, C_d, m*sizeof(int), hipMemcpyDeviceToHost);

	unmap(A);
	unmap(B);
	unmap(C);
	hipFree(A_d); hipFree(B_d); hipFree(C_d);

	dot(A,B,D,m,n);

	for( i = 0; i < m; i++ ) {
		if( C[i] != D[i] ) {
			printf("Error at index %i\n", i);
			return 0;
		}
	}

	free(A); free(B); free(C); free(D);

	printf("Program finished sucessfully.\n");
	return 0;

}
